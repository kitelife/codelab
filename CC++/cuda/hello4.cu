
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  printf("Hello World from block %d and thread %d!\n", bid, tid);
}

int main(void)
{
  hello_from_gpu<<<2, 4>>>();
  hipDeviceSynchronize();
  return 0;
}